#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


torch::Tensor cuda_add(
		torch::Tensor a,
		torch::Tensor b)
{
	auto c = torch::add(a, b);
	return c;
}
